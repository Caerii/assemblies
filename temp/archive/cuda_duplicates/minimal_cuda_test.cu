#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// Simple test kernel
__global__ void test_kernel(float* data, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        data[idx] = data[idx] * 2.0f + 1.0f;
    }
}

// C interface
extern "C" {
    int test_cuda() {
        std::cout << "🧪 Testing CUDA..." << std::endl;
        
        // Allocate device memory
        float* d_data;
        hipMalloc(&d_data, 1024 * sizeof(float));
        
        // Initialize host data
        std::vector<float> h_data(1024, 1.0f);
        
        // Copy to device
        hipMemcpy(d_data, h_data.data(), 1024 * sizeof(float), hipMemcpyHostToDevice);
        
        // Launch kernel
        dim3 block(256);
        dim3 grid((1024 + block.x - 1) / block.x);
        test_kernel<<<grid, block>>>(d_data, 1024);
        
        // Wait for completion
        hipDeviceSynchronize();
        
        // Copy back
        hipMemcpy(h_data.data(), d_data, 1024 * sizeof(float), hipMemcpyDeviceToHost);
        
        // Verify result
        if (h_data[0] == 3.0f) {
            std::cout << "✅ CUDA test passed!" << std::endl;
            hipFree(d_data);
            return 1;
        } else {
            std::cout << "❌ CUDA test failed!" << std::endl;
            hipFree(d_data);
            return 0;
        }
    }
}
