#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <math.h>

namespace assemblies {
namespace cuda {

// Internal kernel implementations

// Weight accumulation kernel - THE HOTTEST KERNEL
__global__ void accumulate_weights_kernel(
    const uint32_t* activated_neurons,     // Input: active neuron IDs
    const float* synapse_weights,          // Input: synapse weights
    const uint32_t* synapse_indices,       // Input: synapse target indices
    const uint32_t* synapse_offsets,       // Input: synapse offsets (CSR format)
    float* activations,                    // Output: accumulated weights
    uint32_t num_activated,                // Input: number of active neurons
    uint32_t target_size                   // Input: target area size
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_activated) return;
    
    uint32_t neuron = activated_neurons[idx];
    uint32_t start = synapse_offsets[neuron];
    uint32_t end = synapse_offsets[neuron + 1];
    
    // Each thread processes synapses for one active neuron
    // This is the most parallelizable part of the algorithm
    for (uint32_t i = start; i < end; i++) {
        uint32_t target = synapse_indices[i];
        float weight = synapse_weights[i];
        
        // Atomic add for thread safety
        atomicAdd(&activations[target], weight);
    }
}

// Parallel Top-K selection using radix selection
__global__ void top_k_selection_kernel(
    const float* activations,              // Input: activation scores
    uint32_t* top_k_indices,              // Output: top-k neuron indices
    uint32_t total_neurons,               // Input: total neurons
    uint32_t k                            // Input: k value
) {
    // Shared memory for local sorting
    extern __shared__ float shared_scores[];
    extern __shared__ uint32_t shared_indices[];
    
    uint32_t tid = threadIdx.x;
    uint32_t bid = blockIdx.x;
    uint32_t block_size = blockDim.x;
    
    // Each block processes a chunk of neurons
    uint32_t start = bid * block_size;
    uint32_t end = min(start + block_size, total_neurons);
    
    // Load data into shared memory
    if (start + tid < end) {
        shared_scores[tid] = activations[start + tid];
        shared_indices[tid] = start + tid;
    } else {
        shared_scores[tid] = -INFINITY;
        shared_indices[tid] = UINT32_MAX;
    }
    
    __syncthreads();
    
    // Parallel bitonic sort for top-k selection
    for (uint32_t i = 0; i < k && i < block_size; i++) {
        for (uint32_t j = tid; j < block_size - 1 - i; j += block_size) {
            if (shared_scores[j] < shared_scores[j + 1]) {
                // Swap scores
                float temp_score = shared_scores[j];
                shared_scores[j] = shared_scores[j + 1];
                shared_scores[j + 1] = temp_score;
                
                // Swap indices
                uint32_t temp_idx = shared_indices[j];
                shared_indices[j] = shared_indices[j + 1];
                shared_indices[j + 1] = temp_idx;
            }
        }
        __syncthreads();
    }
    
    // Store top-k results
    if (tid < k && start + tid < end) {
        top_k_indices[bid * k + tid] = shared_indices[tid];
    }
}

// Parallel candidate generation using truncated normal distribution
__global__ void generate_candidates_kernel(
    hiprandState* states,                   // Input: CUDA RNG states
    float* candidate_weights,             // Output: candidate weights
    uint32_t num_candidates,              // Input: number of candidates
    float mean, float stddev, float cutoff // Input: distribution parameters
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_candidates) return;
    
    hiprandState local_state = states[idx];
    
    // Generate truncated normal samples in parallel
    float sample;
    do {
        sample = hiprand_normal(&local_state) * stddev + mean;
    } while (sample < cutoff);
    
    candidate_weights[idx] = fminf(mean * 2.0f, roundf(sample));
    states[idx] = local_state;
}

// Parallel synapse generation using geometric distribution
__global__ void generate_synapses_kernel(
    hiprandState* states,                   // Input: CUDA RNG states
    uint32_t* synapse_indices,            // Output: synapse target indices
    float* synapse_weights,               // Output: synapse weights
    uint32_t* synapse_offsets,            // Output: synapse offsets (CSR)
    uint32_t support,                     // Input: support size
    float p                               // Input: connection probability
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= support) return;
    
    hiprandState local_state = states[idx];
    
    // Generate synapses using geometric distribution
    uint32_t offset = synapse_offsets[idx];
    uint32_t count = 0;
    
    // Sample from geometric(p) distribution
    float scale = 1.0f / logf(1.0f - p);
    uint32_t last = (uint32_t)floorf(logf(hiprand_uniform(&local_state)) * scale);
    
    while (last < support && offset + count < synapse_offsets[idx + 1]) {
        synapse_indices[offset + count] = last;
        synapse_weights[offset + count] = 1.0f;
        count++;
        
        last += 1 + (uint32_t)floorf(logf(hiprand_uniform(&local_state)) * scale);
    }
    
    states[idx] = local_state;
}

// Parallel plasticity update using Hebbian learning
__global__ void update_plasticity_kernel(
    float* synapse_weights,                // Input/Output: synapse weights
    const uint32_t* activated_neurons,    // Input: activated neuron indices
    const uint32_t* synapse_indices,      // Input: synapse target indices
    const uint32_t* synapse_offsets,      // Input: synapse offsets
    float learn_rate,                     // Input: learning rate
    float max_weight,                     // Input: maximum weight
    uint32_t num_activated                // Input: number of activated neurons
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num_activated) return;
    
    uint32_t neuron = activated_neurons[idx];
    uint32_t start = synapse_offsets[neuron];
    uint32_t end = synapse_offsets[neuron + 1];
    
    // Apply Hebbian learning rule: w += learn_rate * (1 - w)
    for (uint32_t i = start; i < end; i++) {
        float current_weight = synapse_weights[i];
        float new_weight = current_weight + learn_rate * (1.0f - current_weight);
        synapse_weights[i] = fminf(new_weight, max_weight);
    }
}

// Utility kernel for memory initialization
__global__ void initialize_memory_kernel(
    float* data,
    uint32_t size,
    float value
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = value;
    }
}

// Utility kernel for data copying
__global__ void copy_data_kernel(
    const uint32_t* src,
    uint32_t* dst,
    uint32_t size
) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = src[idx];
    }
}

// CUDA random state setup kernel
__global__ void curandSetupKernel(hiprandState* states, unsigned long seed, uint32_t n) {
    uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        hiprand_init(seed, idx, 0, &states[idx]);
    }
}

} // namespace cuda
} // namespace assemblies

// C interface for Python binding
extern "C" {
    
    // C wrapper functions that call the C++ kernels
    __declspec(dllexport) void cuda_accumulate_weights(
        const uint32_t* activated_neurons,
        const float* synapse_weights,
        const uint32_t* synapse_indices,
        const uint32_t* synapse_offsets,
        float* activations,
        uint32_t num_activated,
        uint32_t target_size
    ) {
        // Launch kernel with optimal block size
        dim3 blockSize(256);
        dim3 gridSize((num_activated + blockSize.x - 1) / blockSize.x);
        
        assemblies::cuda::accumulate_weights_kernel<<<gridSize, blockSize>>>(
            activated_neurons, synapse_weights, synapse_indices, synapse_offsets,
            activations, num_activated, target_size
        );
        hipDeviceSynchronize();
    }
    
    __declspec(dllexport) void cuda_generate_candidates(
        hiprandState* states,
        float* candidates,
        uint32_t num_candidates,
        float mean,
        float stddev,
        float cutoff
    ) {
        dim3 blockSize(256);
        dim3 gridSize((num_candidates + blockSize.x - 1) / blockSize.x);
        
        assemblies::cuda::generate_candidates_kernel<<<gridSize, blockSize>>>(
            states, candidates, num_candidates, mean, stddev, cutoff
        );
        hipDeviceSynchronize();
    }
    
    __declspec(dllexport) void cuda_top_k_selection(
        const float* activations,
        uint32_t* top_k_indices,
        uint32_t total_neurons,
        uint32_t k
    ) {
        dim3 blockSize(256);
        dim3 gridSize((total_neurons + blockSize.x - 1) / blockSize.x);
        
        // Calculate shared memory size
        size_t shared_mem_size = blockSize.x * (sizeof(float) + sizeof(uint32_t));
        
        assemblies::cuda::top_k_selection_kernel<<<gridSize, blockSize, shared_mem_size>>>(
            activations, top_k_indices, total_neurons, k
        );
        hipDeviceSynchronize();
    }
    
    __declspec(dllexport) void cuda_initialize_curand(
        hiprandState* states,
        uint32_t n,
        uint32_t seed
    ) {
        dim3 blockSize(256);
        dim3 gridSize((n + blockSize.x - 1) / blockSize.x);
        
        assemblies::cuda::curandSetupKernel<<<gridSize, blockSize>>>(
            states, seed, n
        );
        hipDeviceSynchronize();
    }
    
} // extern "C"
